#include "hip/hip_runtime.h"
/* Copyright (C) 2014 Max Riechelmann <max.riechelmann@googlemail.com>
   (Karlsruhe Institute of Technology)

   This library is free software; you can redistribute it and/or modify it
   under the terms of the GNU Lesser General Public License as published by the
   Free Software Foundation; either version 2.1 of the License, or (at your
   option) any later version.

   This library is distributed in the hope that it will be useful, but WITHOUT
   ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
   FOR A PARTICULAR PURPOSE. See the GNU Lesser General Public License for more
   details.

   You should have received a copy of the GNU Lesser General Public License along
   with this library; if not, write to the Free Software Foundation, Inc., 51
   Franklin St, Fifth Floor, Boston, MA 02110, USA
 */

/**
 * SECTION: test-proxy-gpudirect
 * @short_description: KIRO GPUDIRECT test proxy
 * @title: GPUDIRECTproxy
 * @filename: test-proxy-gpudirect.c
 *
 * GPUDIRECTclient receives data from infiniband, runs one or multiple cuda 
 * kernels on the data and provides the data via server. Receiving and
 * serving data both work via GPUDirect.
 * 
 **/

#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <glib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include "kiro-client.h"
#include "kiro-server.h"

#include "kernels/identity.cu"


/**
 * main
 *
 * This is the main function which connects to the GPUDIRECT server and reads
 * its data via GPUdirect to the GPUs memory. Then it runs a cuda kernel on
 * the gpu to access that memory. Eventually it provides a server to proxy the 
 * data.
 *
 * Run this from shell with kiro-proxy-gpudirect.
 *
 **/
    int 
main ( int argc, char *argv[])
{
    hipError_t error;
    unsigned long int current_frame = 0;
    unsigned long int remote_frame = 0;

    if (argc < 3) {
        g_message ("Not enough arguments. Usage: kiro-proxy-gpudirect <address> <port>\n");
        return -1;
    }

    // Switch on GPU memory allocation and gpudirect data path.
    gpudirect = 1;
    // Select first graphics card.
    hipSetDevice (0);

    // Setup connection.
    KiroClient *kiroClient = kiro_client_new ();
    if (-1 == kiro_client_connect (kiroClient, argv[1], argv[2])) {
        kiro_client_free (kiroClient);
        return -1;
    }

    // Malloc some cuda memory for the kernel result.
    void *result;
    size_t result_size = kiro_client_get_memory_size (kiroClient);
    error = hipMalloc (&result, result_size);
    if (error != 0) {
        g_message ("hipMalloc: %s \n", hipGetErrorString(error));
        return -1;
    }

    // Start the server with that memory.
    KiroServer *kiroServer = kiro_server_new (); 
    if (0 > kiro_server_start (kiroServer, NULL, "60011", result, result_size)) {
        g_critical ("Failed to start server properly");
        goto done;
    }   

    // Now endlessly receive data, run the kernel on it and serve it.
    while (1) {
        // Receive current_frame.
        kiro_client_sync_partial (kiroClient, 0, sizeof (remote_frame), 0);
        error = hipMemcpy (&remote_frame, kiro_client_get_memory (kiroClient), sizeof (remote_frame), hipMemcpyDeviceToHost);
        if (error != 0) {
            g_message ("hipMemcpy: %s \n", hipGetErrorString(error));
            return -1;
        }
        hipDeviceSynchronize ();
        // Check if new data (e.g. new Image) is ready.
        if (remote_frame > current_frame) {
            // Tell user if frames have been skipped.
            if (remote_frame - current_frame - 1) {
                g_warning ("Frames have been skipped! Now at frame: %ld, skipped %ld previous frame(s).", \
                 remote_frame, remote_frame - current_frame - 1);
            }
            // Update current_frame counter.
            current_frame = remote_frame;
            g_warning ("Current Frame: %ld", current_frame);
            // Receive data.
            // TODO: Only sync data when in triple buffering
            kiro_client_sync (kiroClient);
            // Run kernel on data.
            identity <<<1, 1>>> (kiro_client_get_memory (kiroClient), kiro_client_get_memory_size (kiroClient), result, result_size);
            // Wait for kernel to finish.
            hipDeviceSynchronize ();

            // Sleep random amount of time.
            sleep (rand() % 10 / 2);
        }
    }
done:
    kiro_server_free (kiroServer);
    return 0;
}
