
#include <hip/hip_runtime.h>
/**
 * double
 *
 * This is a very simple CUDA kernel that doubles the integers from one gpu memory region
 * and writes the new integers to another gpu memory region.
 *
 **/
__global__
void twice (void *input, int input_size, void *output, int output_size)
{
    // Normal algorithm here.
    int index = (blockIdx.x * blockDim.x + threadIdx.x) + sizeof (long unsigned int);
    ((int *)output)[index/4] = 2*((int *)input)[index/4];

    // Increment frame. This always has to happen last, so the client knows, it's ready.
    *(unsigned long int *)output = *(unsigned long int *)input;

}
