
#include <hip/hip_runtime.h>
/**
 * indentity
 *
 * This is a very simple CUDA kernel that copies the data from one gpu memory region
 * to another.
 *
 **/
__global__
void identity (void *input, int input_size, void *output, int output_size)
{
    // Normal algorithm here.
    int index = (blockIdx.x * blockDim.x + threadIdx.x) + sizeof (long unsigned int);
    ((int *)output)[index/4] = ((int *)input)[index/4];

    // Increment frame. This always has to happen last, so the client knows, it's ready.
    *(unsigned long int *)output = *(unsigned long int *)input;

}
